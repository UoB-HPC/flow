#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "../wet.h"
#include "../../comms.h"

#define set_cuda_indices(padx) \
  const int gid = threadIdx.x+blockIdx.x*blockDim.x; \
const int jj = (gid % (nx+padx));\
const int ii = (gid / (nx+padx));

#define ind0 (ii*nx + jj)
#define ind1 (ii*(nx+1) + jj)

// Solve a single timestep on the given mesh
void solve_hydro(
    Mesh* mesh, int tt, double* P, double* rho, double* rho_old, 
    double* e, double* u, double* v, double* rho_u, double* rho_v, 
    double* Qxx, double* Qyy, double* F_x, double* F_y, double* uF_x, 
    double* uF_y, double* vF_x, double* vF_y, double* min_timesteps)
{
  if(mesh->rank == MASTER)
    printf("dt %.12e dt_h %.12e\n", mesh->dt, mesh->dt_h);

  int nthreads_per_block = ceil(mesh->local_nx*mesh->local_ny/(double)NBLOCKS);
  equation_of_state<<<nthreads_per_block, NBLOCKS>>>(
      mesh->local_nx, mesh->local_ny, P, rho, e);

  nthreads_per_block = ceil((mesh->local_nx+1)*(mesh->local_ny+1)/(double)NBLOCKS);
  pressure_acceleration<<<nthreads_per_block, NBLOCKS>>>(
      mesh->local_nx, mesh->local_ny, mesh, mesh->dt, rho_u, rho_v, 
      u, v, P, rho, mesh->edgedx, mesh->edgedy, 
      mesh->celldx, mesh->celldy);

  handle_boundary(nx+1, ny, mesh, u, INVERT_X, PACK);
  handle_boundary(nx, ny+1, mesh, v, INVERT_Y, PACK);

  artificial_viscosity(
      mesh->local_nx, mesh->local_ny, mesh, mesh->dt, Qxx, Qyy, 
      u, v, rho_u, rho_v, rho, 
      mesh->edgedx, mesh->edgedy, mesh->celldx, mesh->celldy);

  nthreads_per_block = ceil(mesh->local_nx*mesh->local_ny/(double)NBLOCKS);
  shock_heating_and_work(
      mesh->local_nx, mesh->local_ny, mesh, mesh->dt_h, e, P, u, 
      v, rho, Qxx, Qyy, mesh->celldx, mesh->celldy);

  handle_boundary(nx, ny, mesh, e, NO_INVERT, PACK);

  set_timestep(
      mesh->local_nx, mesh->local_ny, Qxx, Qyy, rho, 
      e, mesh, min_timesteps, tt == 0, mesh->celldx, mesh->celldy);

  // Perform advection
  advect_mass_and_energy(
      mesh->local_nx, mesh->local_ny, mesh, tt, mesh->dt, mesh->dt_h, rho, e, rho_old, F_x, F_y, 
      uF_x, uF_y, u, v, mesh->edgedx, mesh->edgedy, mesh->celldx, mesh->celldy);

  advect_momentum(
      mesh->local_nx, mesh->local_ny, tt, mesh, mesh->dt_h, mesh->dt, u, v, 
      uF_x, uF_y, vF_x, vF_y, rho_u, rho_v, rho, F_x, F_y, 
      mesh->edgedx, mesh->edgedy, mesh->celldx, mesh->celldy);
}

// Calculate the pressure from GAMma law equation of state
__global__ void equation_of_state(
    const int nx, const int ny, double* P, const double* rho, const double* e)
{
  set_cuda_indices(0);

  // Only invoke simple GAMma law at the moment
  P[ind0] = (GAM - 1.0)*rho[ind0]*e[ind0];
}

// Calculate change in momentum caused by pressure gradients, and then extract
// the velocities using edge centered density approximations
void pressure_acceleration(
    const int nx, const int ny, Mesh* mesh, const double dt, double* rho_u, 
    double* rho_v, double* u, double* v, const double* P, const double* rho,
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(1);

  if(ii < PAD || jj < PAD || jj >= (nx+1)-PAD || ii >= (ny+1)-PAD) 
    return;

  // Update the momenta using the pressure gradients
  rho_u[ind1] -= dt*(P[ind0] - P[ind0-1])/edgedx[jj];
  rho_v[ind0] -= dt*(P[ind0] - P[ind0-nx])/edgedy[ii];

  // Calculate the zone edge centered density
  const double rho_edge_x = 
    (rho[ind0]*celldx[jj]*celldy[ii] + rho[ind0-1]*celldx[jj - 1]*celldy[ii])/ 
    (2.0*edgedx[jj]*celldy[ii]);
  const double rho_edge_y = 
    (rho[ind0]*celldx[jj]*celldy[ii] + rho[ind0-nx]*celldx[jj]*celldy[ii - 1])/ 
    (2.0*celldx[jj]*edgedy[ii]);

  // Find the velocities from the momenta and edge centered mass densities
  u[ind1] = (rho_edge_x == 0.0) ? 0.0 : rho_u[ind1] / rho_edge_x;
  v[ind0] = (rho_edge_y == 0.0) ? 0.0 : rho_v[ind0] / rho_edge_y;
}

void artificial_viscosity(
    const int nx, const int ny, Mesh* mesh, const double dt, double* Qxx, 
    double* Qyy, double* u, double* v, double* rho_u, double* rho_v, const double* rho, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  int nthreads_per_block = ceil(nx*ny/(double)NBLOCKS);
  calc_viscous_stresses<<<nthreads_per_block, NBLOCKS>>>(
      nx, ny, mesh, dt, Qxx, Qyy, u, v, rho_u, rho_v, rho, 
      edgedx, edgedy, celldx, celldy);

  handle_boundary(nx, ny, mesh, Qxx, NO_INVERT, PACK);
  handle_boundary(nx, ny, mesh, Qyy, NO_INVERT, PACK);

  nthreads_per_block = ceil((nx+1)*(ny+1)/(double)NBLOCKS);
  viscous_acceleration(
      nx, ny, mesh, dt, Qxx, Qyy, u, v, rho_u, rho_v, rho, 
      edgedx, edgedy, celldx, celldy);

  handle_boundary(nx+1, ny, mesh, u, INVERT_X, PACK);
  handle_boundary(nx, ny+1, mesh, v, INVERT_Y, PACK);
}

__global__ void calc_viscous_stresses(
    const int nx, const int ny, Mesh* mesh, const double dt, double* Qxx, 
    double* Qyy, double* u, double* v, double* rho_u, double* rho_v, const double* rho, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= ny-PAD) 
    return;

  const double u_i = min(0.0, u[ind1+1] - u[ind1]);
  const double u_ii = 0.5*(
      fabs(min(0.0, (u[ind1+2]-u[ind1+1])) - min(0.0, (u[ind1+1]-u[ind1]))) + 
      fabs(min(0.0, (u[ind1+1]-u[ind1])) - min(0.0, (u[ind1]-u[ind1-1]))));
  const double v_i = min(0.0, v[ind0+nx] - v[ind0]);
  const double v_ii = 0.5*(
      fabs(min(0.0, (v[ind0+2*nx]-v[ind0+nx])) - min(0.0, (v[ind0+nx]-v[ind0]))) + 
      fabs(min(0.0, (v[ind0+nx]-v[ind0])) - min(0.0, (v[ind0]-v[ind0-nx]))));
  Qxx[ind0] = -C_Q*rho[ind0]*u_i*u_ii;
  Qyy[ind0] = -C_Q*rho[ind0]*v_i*v_ii;
}

__global__ void viscous_acceleration(
    const int nx, const int ny, Mesh* mesh, const double dt, double* Qxx, 
    double* Qyy, double* u, double* v, double* rho_u, double* rho_v, const double* rho, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(1);

  if(ii < PAD || jj < PAD || jj >= (nx+1)-PAD || ii >= (ny+1)-PAD) 
    return;

  rho_u[ind1] -= dt*(Qxx[ind0] - Qxx[ind0-1])/celldx[jj];
  rho_v[ind0] -= dt*(Qyy[ind0] - Qyy[ind0-nx])/celldy[ii];

  // Calculate the zone edge centered density
  const double rho_edge_x = 
    (rho[ind0]*celldx[jj]*celldy[ii] + rho[ind0-1]*celldx[jj-1]*celldy[ii]) / 
    (2.0*edgedx[jj]*celldy[ii]);
  const double rho_edge_y = 
    (rho[ind0]*celldx[jj]*celldy[ii] + rho[ind0-nx]*celldx[jj]*celldy[ii-1]) / 
    (2.0*celldx[jj]*edgedy[ii]);

  // Find the velocities from the momenta and edge centered mass densities
  u[ind1] = (rho_edge_x == 0.0) ? 0.0 : rho_u[ind1] / rho_edge_x;
  v[ind0] = (rho_edge_y == 0.0) ? 0.0 : rho_v[ind0] / rho_edge_y;
}

// Calculates the work done due to forces within the element
__global__ void shock_heating_and_work(
    const int nx, const int ny, Mesh* mesh, const double dt_h, double* e, 
    const double* P, const double* u, const double* v, const double* rho, 
    const double* Qxx, const double* Qyy, const double* celldx, const double* celldy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= ny-PAD) 
    return;

  const double div_vel_x = (u[ind1+1] - u[ind1])/celldx[jj];
  const double div_vel_y = (v[ind0+nx] - v[ind0])/celldy[ii];
  const double div_vel_dt = (div_vel_x + div_vel_y)*dt_h;
  const double e_q = e[ind0] - dt_h*(Qxx[ind0]*div_vel_x + Qyy[ind0]*div_vel_y)/rho[ind0];

  /// A working formulation that is second order in time for Pressure!?
  const double rho_c = rho[ind0]/(1.0 + div_vel_dt);
  const double e_c = e_q - (P[ind0]*div_vel_dt)/rho[ind0];
  const double work = 0.5*div_vel_dt*(P[ind0] + (GAM-1.0)*e_c*rho_c)/rho[ind0];
  e[ind0] = (rho[ind0] == 0.0) ? 0.0 : e_q-work;
}

// Calculates the timestep from the current state
void set_timestep(
    const int nx, const int ny, double* Qxx, double* Qyy, const double* rho, 
    const double* e, Mesh* mesh, double* min_timesteps, const int first_step,
    const double* celldx, const double* celldy)
{
  double local_min_dt = MAX_DT;

  int nthreads_per_block = ceil((nx+1)*(ny+1)/(double)NBLOCKS);
  calc_min_timestep<NBLOCKS><<<nthreads_per_block, NBLOCKS>>>(
      nx, ny, Qxx, Qyy, rho, e, Mesh* mesh, min_timesteps, first_step, celldx, celldy);

  // TODO: This is not right, it doesn't reduce all values
  while(nthreads_per_block > 1) {
    min_reduce<NBLOCKS>(min_timesteps, min_timesteps);
    nthreads_per_block = ceil(nthreads_per_block/(double)NBLOCKS);
  }

  sync_data(1, 1, min_timesteps, RECV);

  // Ensure that the timestep does not jump too far from one step to the next
  double global_min_dt = reduce_all_min(min_timesteps[0]);
  const double final_min_dt = min(global_min_dt, C_M*mesh->dt_h);
  mesh->dt = 0.5*(C_T*final_min_dt + mesh->dt_h);
  mesh->dt_h = (first_step) ? mesh->dt : C_T*final_min_dt;
}

  template <unsigned int block_size>
__global__ void calc_min_timestep(
    const int nx, const int ny, double* Qxx, double* Qyy, const double* rho, 
    const double* e, Mesh* mesh, double* min_timesteps, const int first_step,
    const double* celldx, const double* celldy)
{
  // Constrain based on the sound speed within the system
  const double c_s = sqrt(GAM*(GAM - 1.0)*e[ind0]);
  const double thread_min_dt_x = celldx[jj]/sqrt(c_s*c_s + 2.0*Qxx[ind0]/rho[ind0]);
  const double thread_min_dt_y = celldy[ii]/sqrt(c_s*c_s + 2.0*Qyy[ind0]/rho[ind0]);
  const double thread_min_dt = min(thread_min_dt_x, thread_min_dt_y);

  __shared__ sdata[block_size];
  const int tid = threadIdx.x;
  sdata[tid] = min(local_min_dt, thread_min_dt);
  __syncthreads();

  min_reduce_in_shared<NBLOCKS>(threadIdx, sdata);
  if (tid == 0) result[blockIdx.x] = sdata[0];
}

// Perform advection with monotonicity improvement
void advect_mass_and_energy(
    const int nx, const int ny, Mesh* mesh, const int tt, const double dt,
    const double dt_h, double* rho, double* e, double* rho_old, double* F_x, double* F_y, 
    double* eF_x, double* eF_y, const double* u, const double* v, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  int nthreads_per_block = ceil(nx*ny/(double)NBLOCKS);
  store_old_rho<<<nthreads_per_block, NBLOCKS>>>(rho, rho_old);

  if(tt % 2 == 0) {
    mass_and_energy_x_advection(
        nx, ny, 1, mesh, dt, dt_h, rho, rho_old, e, u, F_x, eF_x, 
        celldx, edgedx, celldy, edgedy);
    mass_and_energy_y_advection(
        nx, ny, 0, mesh, dt, dt_h, rho, rho_old, e, v, F_y, eF_y, 
        celldx, edgedx, celldy, edgedy);
  }
  else {
    mass_and_energy_y_advection(
        nx, ny, 1, mesh, dt, dt_h, rho, rho_old, e, v, F_y, eF_y, 
        celldx, edgedx, celldy, edgedy);
    mass_and_energy_x_advection(
        nx, ny, 0, mesh, dt, dt_h, rho, rho_old, e, u, F_x, eF_x, 
        celldx, edgedx, celldy, edgedy);
  }
}

__global__ void store_old_rho(
    double* rho, double* rho_old)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= ny-PAD) 
    return;

  rho_old[ind0] = rho[ind0];
}

// Advect energy and mass in the x direction
void mass_and_energy_x_advection(
    const int nx, const int ny, const int first, Mesh* mesh, const double dt, 
    const double dt_h, double* rho, double* rho_old, double* e, const double* u, 
    double* F_x, double* eF_x, const double* celldx, const double* edgedx, 
    const double* celldy, const double* edgedy)
{
  int nthreads_per_block = ceil((nx+1)*ny/(double)NBLOCKS);
  calc_x_mass_and_energy_flux<<<nthreads_per_block, NBLOCKS>>>(
      nx, ny, first, mesh, dt, dt_h, rho, rho_old, e, u, 
      F_x, eF_x, celldx, edgedx, celldy, edgedy);

  handle_boundary(nx+1, ny, mesh, F_x, INVERT_X, PACK);

  nthreads_per_block = ceil(nx*ny/(double)NBLOCKS);
  advect_mass_and_energy_in_x<<<nthreads_per_block, NBLOCKS>>>(
      nx, ny, first, mesh, dt, dt_h, rho, rho_old, e, u, 
      F_x, eF_x, celldx, edgedx, celldy, edgedy);

  handle_boundary(nx, ny, mesh, rho, NO_INVERT, PACK);
  handle_boundary(nx, ny, mesh, e, NO_INVERT, PACK);
}

// Calculate the flux in the x direction
void calc_x_mass_and_energy_flux(
    const int nx, const int ny, const int first, Mesh* mesh, const double dt, 
    const double dt_h, double* rho, double* rho_old, double* e, const double* u, 
    double* F_x, double* eF_x, const double* celldx, const double* edgedx, 
    const double* celldy, const double* edgedy)
{
  set_cuda_indices(1);

  if(ii < PAD || jj < PAD || jj >= (nx+1)-PAD || ii >= ny-PAD) 
    return;

  // Interpolate to make second order in time
  const double invdx = 1.0/edgedx[jj];
  const double suc0 = 0.5*invdx*(u[ind1+1]-u[ind1-1]);
  const double sur0 = 2.0*invdx*(u[ind1]-u[ind1-1]);
  const double sul0 = 2.0*invdx*(u[ind1+1]-u[ind1]);
  const double u_tc = u[ind1] - 0.5*u[ind1]*dt*minmod(suc0, minmod(sur0, sul0));

  // Van leer limiter
  double limiter = 0.0;
  const double rho_diff = (rho[ind0]-rho[ind0-1]);
  if(rho_diff) {
    const double smoothness = (u_tc >= 0.0) 
      ? (rho[ind0-1]-rho[ind0-2])/rho_diff
      : (rho[ind0+1]-rho[ind0])/rho_diff;
    limiter = (smoothness + fabs(smoothness))/(1.0+fabs(smoothness));
  }

  // Calculate the flux
  const double rho_upwind = (u_tc >= 0.0) ? rho[ind0-1] : rho[ind0];
  F_x[ind1] = (u_tc*rho_upwind+
      0.5*fabs(u_tc)*(1.0-fabs((u_tc*dt_h)/celldx[jj]))*limiter*rho_diff);

  // Use MC limiter to get slope of energy
  const double a_x_0 = 0.5*invdx*(e[ind0]-e[ind0-2]);
  const double b_x_0 = 2.0*invdx*(e[ind0-1]-e[ind0-2]);
  const double c_x_0 = 2.0*invdx*(e[ind0]-e[ind0-1]);
  const double a_x_1 = 0.5*invdx*(e[ind0+1]-e[ind0-1]);
  const double b_x_1 = 2.0*invdx*(e[ind0]-e[ind0-1]);
  const double c_x_1 = 2.0*invdx*(e[ind0+1]-e[ind0]);

  // Calculate the interpolated densities
  const double edge_e_x = (u_tc > 0.0)
    ? e[ind0-1] + 0.5*minmod(minmod(a_x_0, b_x_0), c_x_0)*(celldx[jj-1] - u_tc*dt_h)
    : e[ind0] - 0.5*minmod(minmod(a_x_1, b_x_1), c_x_1)*(celldx[jj] + u_tc*dt_h);

  // Update the fluxes to now include the contribution from energy
  eF_x[ind1] = edgedy[ii]*edge_e_x*F_x[ind1]; 
}

// Advect mass and energy in the x direction
void advect_mass_and_energy_in_x(
    const int nx, const int ny, const int first, Mesh* mesh, const double dt, 
    const double dt_h, double* rho, double* rho_old, double* e, const double* u, 
    double* F_x, double* eF_x, const double* celldx, const double* edgedx, 
    const double* celldy, const double* edgedy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= ny-PAD) 
    return;

  rho[ind0] -= dt_h*
    (edgedy[ii+1]*F_x[ind1+1] - edgedy[ii]*F_x[ind1])/ 
    (celldx[jj]*celldy[ii]);
  const double rho_e = (rho_old[ind0]*e[ind0] - 
      (dt_h*(eF_x[ind1+1] - eF_x[ind1]))/(celldx[jj]*celldy[ii]));
  e[ind0] = (first) 
    ? (rho_old[ind0] == 0.0) ? 0.0 : rho_e/rho_old[ind0]
    : (rho[ind0] == 0.0) ? 0.0 : rho_e/rho[ind0];
}

// Advect energy and mass in the x direction
void mass_and_energy_x_advection(
    const int nx, const int ny, const int first, Mesh* mesh, const double dt,
    const double dt_h, double* rho, double* rho_old, double* e, const double* v, 
    double* F_y, double* eF_y, const double* celldx, const double* edgedx, 
    const double* celldy, const double* edgedy)
{
  int nthreads_per_block = ceil(nx*(ny+1)/(double)NBLOCKS);
  calc_y_mass_and_energy_flux<<<nthreads_per_block, NBLOCKS>>>(
      nx, ny, first, Mesh* mesh, dt, dt_h, rho, rho_old, e, v, 
      F_y, eF_y, celldx, edgedx, celldy, edgedy);

  handle_boundary(nx, ny+1, mesh, F_y, INVERT_Y, PACK);

  nthreads_per_block = ceil(nx*ny/(double)NBLOCKS);
  advect_mass_and_energy_in_y<<<nthreads_per_block, NBLOCKS>>>(
      nx, ny, first, Mesh* mesh, dt, dt_h, rho, rho_old, e, v, 
      F_y, eF_y, celldx, edgedx, celldy, edgedy);

  handle_boundary(nx, ny, mesh, rho, NO_INVERT, PACK);
  handle_boundary(nx, ny, mesh, e, NO_INVERT, PACK);
}

void calc_y_mass_and_energy_flux(
    const int nx, const int ny, const int first, Mesh* mesh, const double dt,
    const double dt_h, double* rho, double* rho_old, double* e, const double* v, 
    double* F_y, double* eF_y, const double* celldx, const double* edgedx, 
    const double* celldy, const double* edgedy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= (ny+1)-PAD) 
    return;

  // Compute the mass flux along the y edges
  // In the ghost cells flux is left as 0.0
  // Interpolate the velocity to make second order in time
  const double invdy = 1.0/edgedy[ii];
  const double svc0 = 0.5*invdy*(v[ind0+nx]-v[ind0-nx]);
  const double svr0 = 2.0*invdy*(v[ind0]-v[ind0-nx]);
  const double svl0 = 2.0*invdy*(v[ind0+nx]-v[ind0]);
  const double v_tc = v[ind0] - 0.5*v[ind0]*dt*minmod(svc0, minmod(svr0, svl0));

  // Van leer limiter
  const double rho_diff = (rho[ind0]-rho[ind0-nx]);
  double limiter = 0.0;
  if(rho_diff) {
    const double smoothness = (v[ind0] >= 0.0) 
      ? (rho[ind0-nx]-rho[ind0-2*nx])/rho_diff
      : (rho[ind0+nx]-rho[ind0])/rho_diff;
    limiter = (smoothness + fabs(smoothness))/(1.0+fabs(smoothness));
  }

  // Calculate the flux
  const double rho_upwind = (v_tc >= 0.0) ? rho[ind0-nx] : rho[ind0];
  F_y[ind0] = (v_tc*rho_upwind+
      0.5*fabs(v_tc)*(1.0-fabs((v_tc*dt_h)/celldy[ii]))*limiter*rho_diff);

  // Use MC limiter to get slope of energy
  const double a_y_0 = 0.5*invdy*(e[ind0]-e[ind0-2*nx]);
  const double b_y_0 = 2.0*invdy*(e[ind0-nx]-e[ind0-2*nx]);
  const double c_y_0 = 2.0*invdy*(e[ind0]-e[ind0-nx]);
  const double a_y_1 = 0.5*invdy*(e[ind0+nx]-e[ind0-nx]);
  const double b_y_1 = 2.0*invdy*(e[ind0]-e[ind0-nx]);
  const double c_y_1 = 2.0*invdy*(e[ind0+nx]-e[ind0]);

  const double edge_e_y = (v_tc > 0.0)
    ? e[ind0-nx] + 0.5*minmod(minmod(a_y_0, b_y_0), c_y_0)*(celldy[ii-1] - v_tc*dt_h)
    : e[ind0] - 0.5*minmod(minmod(a_y_1, b_y_1), c_y_1)*(celldy[ii] + v_tc*dt_h);

  // Update the fluxes to now include the contribution from energy
  eF_y[ind0] = edgedx[jj]*edge_e_y*F_y[ind0]; 
}

void advect_mass_and_energy_in_y(
    const int nx, const int ny, const int first, Mesh* mesh, const double dt,
    const double dt_h, double* rho, double* rho_old, double* e, const double* v, 
    double* F_y, double* eF_y, const double* celldx, const double* edgedx, 
    const double* celldy, const double* edgedy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= ny-PAD) 
    return;

  rho[ind0] -= dt_h*
    (edgedx[jj+1]*F_y[ind0+nx] - edgedx[jj]*F_y[ind0])/
    (celldx[jj]*celldy[ii]);
  const double rho_e = (rho_old[ind0]*e[ind0] - 
      (dt_h*(eF_y[ind0+nx] - eF_y[ind0]))/(celldx[jj]*celldy[ii]));
  e[ind0] = (first) 
    ? (rho_old[ind0] == 0.0) ? 0.0 : rho_e/rho_old[ind0]
    : (rho[ind0] == 0.0) ? 0.0 : rho_e/rho[ind0];
}

__global__ void ux_momentum_flux(
    const int nx, const int ny, Mesh* mesh, const double dt_h, const double dt, 
    double* u, double* v, double* uF_x, double* rho_u, const double* rho, const double* F_x, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= ny-PAD) 
    return;

  // Calculate the cell centered x momentum fluxes in the x direction
  // Use MC limiter to get slope of velocity
  const double invdx = 1.0/edgedx[jj];
  const double a_x_0 = 0.5*invdx*(u[ind1+1]-u[ind1-1]);
  const double b_x_0 = 2.0*invdx*(u[ind1]-u[ind1-1]);
  const double c_x_0 = 2.0*invdx*(u[ind1+1]-u[ind1]);
  const double a_x_1 = 0.5*invdx*(u[ind1+2]-u[ind1]);
  const double b_x_1 = 2.0*invdx*(u[ind1+1]-u[ind1]);
  const double c_x_1 = 2.0*invdx*(u[ind1+2]-u[ind1+1]);

  // Calculate the interpolated densities
  const double u_cell_x = 0.5*(u[ind1]+u[ind1+1]);
  const double f_x = edgedy[ii]*0.5*(F_x[ind1] + F_x[ind1+1]); 
  const double u_cell_x_interp = (u_cell_x > 0.0)
    ? u[ind1] + 0.5*minmod(minmod(a_x_0, b_x_0), c_x_0)*(celldx[jj-1] - u_cell_x*dt_h)
    : u[ind1+1] - 0.5*minmod(minmod(a_x_1, b_x_1), c_x_1)*(celldx[jj] + u_cell_x*dt_h);
  uF_x[ind0] = f_x*u_cell_x_interp;
}


void advect_rho_u_in_x(
    const int nx, const int ny, const int tt, Mesh* mesh, const double dt_h, 
    const double dt, double* u, double* v, double* uF_x, double* uF_y, 
    double* vF_x, double* vF_y, double* rho_u, double* rho_v, 
    const double* rho, const double* F_x, const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(1);

  if(ii < PAD || jj < PAD || jj >= (nx+1)-PAD || ii >= ny-PAD) 
    return;

  rho_u[ind1] -= dt_h*(uF_x[ind0] - uF_x[ind0-1])/(edgedx[jj]*celldy[ii]);
}

void advect_rho_u_and_u_in_x(
    const int nx, const int ny, const int tt, Mesh* mesh, const double dt_h, 
    const double dt, double* u, double* v, double* uF_x, double* uF_y, 
    double* vF_x, double* vF_y, double* rho_u, double* rho_v, 
    const double* rho, const double* F_x, const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(1);

  if(ii < PAD || jj < PAD || jj >= (nx+1)-PAD || ii >= ny-PAD) 
    return;

  rho_u[ind1] -= dt_h*(uF_x[ind0] - uF_x[ind0-1])/(edgedx[jj]*celldy[ii]);
  const double rho_edge_x = 
    (rho[ind0]*celldx[jj]*celldy[ii] + rho[ind0-1]*celldx[jj - 1]*celldy[ii])/ 
    (2.0*edgedx[jj]*celldy[ii]);
  u[ind1] = (rho_edge_x == 0.0) ? 0.0 : rho_u[ind1] / rho_edge_x;
}

void advect_rho_u_in_y(
    const int nx, const int ny, const int tt, Mesh* mesh, const double dt_h, 
    const double dt, double* u, double* v, double* uF_x, double* uF_y, 
    double* vF_x, double* vF_y, double* rho_u, double* rho_v, 
    const double* rho, const double* F_x, const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(1);

  if(ii < PAD || jj < PAD || jj >= (nx+1)-PAD || ii >= ny-PAD) 
    return;

  rho_u[ind1] -= dt_h*(uF_y[ind1+(nx+1)] - uF_y[ind1])/(celldx[jj]*edgedy[ii]);
}

void advect_rho_u_and_u_in_y(
    const int nx, const int ny, const int tt, Mesh* mesh, const double dt_h, 
    const double dt, double* u, double* v, double* uF_x, double* uF_y, 
    double* vF_x, double* vF_y, double* rho_u, double* rho_v, 
    const double* rho, const double* F_x, const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(1);

  if(ii < PAD || jj < PAD || jj >= (nx+1)-PAD || ii >= ny-PAD) 
    return;

  rho_u[ind1] -= dt_h*(uF_y[ind1+(nx+1)] - uF_y[ind1])/(celldx[jj]*edgedy[ii]);
  const double rho_edge_x = 
    (rho[ind0]*celldx[jj]*celldy[ii] + rho[ind0-1]*celldx[jj - 1]*celldy[ii])/ 
    (2.0*edgedx[jj]*celldy[ii]);
  u[ind1] = (rho_edge_x == 0.0) ? 0.0 : rho_u[ind1] / rho_edge_x;
}

void uy_momentum_flux(
    const int nx, const int ny, Mesh* mesh, const double dt_h, const double dt, 
    double* u, double* v, double* uF_y, double* rho_u, const double* rho, 
    const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(1);

  if(ii < PAD || jj < PAD || jj >= (nx+1)-PAD || ii >= (ny+1)-PAD) 
    return;

  // Use MC limiter to get slope of velocity
  const double invdy = 1.0/edgedy[ii];
  const double a_y_0 = 0.5*invdy*(u[ind1]-u[ind1-2*(nx+1)]);
  const double b_y_0 = 2.0*invdy*(u[ind1-(nx+1)]-u[ind1-2*(nx+1)]);
  const double c_y_0 = 2.0*invdy*(u[ind1]-u[ind1-(nx+1)]);
  const double a_y_1 = 0.5*invdy*(u[ind1+(nx+1)]-u[ind1-(nx+1)]);
  const double b_y_1 = 2.0*invdy*(u[ind1]-u[ind1-(nx+1)]);
  const double c_y_1 = 2.0*invdy*(u[ind1+(nx+1)]-u[ind1]);
  const double v_cell_y = 0.5*(v[ind0-1]+v[ind0]);

  const double f_y = edgedx[jj]*0.5*(F_y[ind0] + F_y[ind0-1]);
  const double u_corner_y = (v_cell_y > 0.0)
    ? u[ind1-(nx+1)] + 0.5*minmod(minmod(a_y_0, b_y_0), c_y_0)*(celldy[ii-1] - v_cell_y*dt_h)
    : u[ind1] - 0.5*minmod(minmod(a_y_1, b_y_1), c_y_1)*(celldy[ii] + v_cell_y*dt_h);
  uF_y[ind1] = f_y*u_corner_y;
}

void vx_momentum_flux(
    const int nx, const int ny, Mesh* mesh, const double dt_h, const double dt, 
    const double* u, double* v, double* vF_x, double* rho_v, const double* rho, 
    const double* F_x, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(1);

  if(ii < PAD || jj < PAD || jj >= (nx+1)-PAD || ii >= (ny+1)-PAD) 
    return;

  // Use MC limiter to get slope of velocity
  const double invdx = 1.0/edgedx[jj];
  const double a_x_0 = 0.5*invdx*(v[ind0]-v[ind0-2]);
  const double b_x_0 = 2.0*invdx*(v[ind0-1]-v[ind0-2]);
  const double c_x_0 = 2.0*invdx*(v[ind0]-v[ind0-1]);
  const double a_x_1 = 0.5*invdx*(v[ind0+1]-v[ind0-1]);
  const double b_x_1 = 2.0*invdx*(v[ind0]-v[ind0-1]);
  const double c_x_1 = 2.0*invdx*(v[ind0+1]-v[ind0]);

  // Calculate the interpolated densities
  const double f_x = celldy[ii]*0.5*(F_x[ind1] + F_x[ind1-(nx+1)]);
  const double u_cell_x = 0.5*(u[ind1]+u[ind1-(nx+1)]);
  const double v_cell_x_interp = (u_cell_x > 0.0)
    ? v[ind0-1] + 0.5*minmod(minmod(a_x_0, b_x_0), c_x_0)*(celldx[jj-1] - u_cell_x*dt_h)
    : v[ind0] - 0.5*minmod(minmod(a_x_1, b_x_1), c_x_1)*(celldx[jj] + u_cell_x*dt_h);
  vF_x[ind1] = f_x*v_cell_x_interp;
}

void advect_rho_v_and_v_in_y(
    const int nx, const int ny, Mesh* mesh, const double dt_h, const double dt, 
    double* u, double* v, double* vF_y, double* rho_v, const double* rho, const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= (ny+1)-PAD) 
    return;

  rho_v[ind0] -= dt_h*(vF_y[ind0] - vF_y[ind0-nx])/(celldx[jj]*edgedy[ii]);
  const double rho_edge_y = 
    (rho[ind0]*celldx[jj]*celldy[ii] + rho[ind0-nx]*celldx[jj]*celldy[ii - 1])/ 
    (2.0*celldx[jj]*edgedy[ii]);
  v[ind0] = (rho_edge_y == 0.0) ? 0.0 : rho_v[ind0] / rho_edge_y;
}

void advect_rho_v_and_v_in_x(
    const int nx, const int ny, Mesh* mesh, const double dt_h, const double dt, 
    const double* u, double* v, double* vF_x, double* rho_v, const double* rho, 
    const double* F_x, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= (ny+1)-PAD) 
    return;

  rho_v[ind0] -= dt_h*(vF_x[ind1+1] - vF_x[ind1])/(edgedx[jj]*celldy[ii]);
  const double rho_edge_y = 
    (rho[ind0]*celldx[jj]*celldy[ii] + rho[ind0-nx]*celldx[jj]*celldy[ii - 1])/ 
    (2.0*celldx[jj]*edgedy[ii]);
  v[ind0] = (rho_edge_y == 0.0) ? 0.0 : rho_v[ind0] / rho_edge_y;
}

void advect_rho_v_in_x(
    const int nx, const int ny, Mesh* mesh, const double dt_h, const double dt, 
    const double* u, double* v, double* vF_x, double* rho_v, const double* rho, 
    const double* F_x, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= (ny+1)-PAD) 
    return;

  rho_v[ind0] -= dt_h*(vF_x[ind1+1] - vF_x[ind1])/(edgedx[jj]*celldy[ii]);
}

void vy_momentum_flux(
    const int nx, const int ny, Mesh* mesh, const double dt_h, const double dt, 
    double* u, double* v, double* vF_y, double* rho_v, const double* rho, const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= ny-PAD) 
    return;

  // Use MC limiter to get slope of velocity
  const double invdy = 1.0/edgedy[ii];
  const double a_y_0 = 0.5*invdy*(v[ind0+nx]-v[ind0-nx]);
  const double b_y_0 = 2.0*invdy*(v[ind0]-v[ind0-nx]);
  const double c_y_0 = 2.0*invdy*(v[ind0+nx]-v[ind0]);
  const double a_y_1 = 0.5*invdy*(v[ind0+2*nx]-v[ind0]);
  const double b_y_1 = 2.0*invdy*(v[ind0+nx]-v[ind0]);
  const double c_y_1 = 2.0*invdy*(v[ind0+2*nx]-v[ind0+nx]);

  const double f_y = celldx[jj]*0.5*(F_y[ind0] + F_y[ind0+nx]);
  const double v_cell_y = 0.5*(v[ind0]+v[ind0+nx]);
  const double v_cell_y_interp = (v_cell_y > 0.0)
    ? v[ind0] + 0.5*minmod(minmod(a_y_0, b_y_0), c_y_0)*(celldy[ii-1] - v_cell_y*dt_h)
    : v[ind0+nx] - 0.5*minmod(minmod(a_y_1, b_y_1), c_y_1)*(celldy[ii] + v_cell_y*dt_h);
  vF_y[ind0] = f_y*v_cell_y_interp;
}

void advect_rho_v_in_y(
    const int nx, const int ny, Mesh* mesh, const double dt_h, const double dt, 
    double* u, double* v, double* vF_y, double* rho_v, const double* rho, const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  set_cuda_indices(0);

  if(ii < PAD || jj < PAD || jj >= nx-PAD || ii >= (ny+1)-PAD) 
    return;

  rho_v[ind0] -= dt_h*(vF_y[ind0] - vF_y[ind0-nx])/(celldx[jj]*edgedy[ii]);
}

// Advect momentum according to the velocity
void advect_momentum(
    const int nx, const int ny, const int tt, Mesh* mesh, const double dt_h, 
    const double dt, double* u, double* v, double* uF_x, double* uF_y, 
    double* vF_x, double* vF_y, double* rho_u, double* rho_v, 
    const double* rho, const double* F_x, const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  int nthreads_per_block = 0;
  if(tt % 2) {
    nthreads_per_block = ceil(nx*ny/(double)NBLOCKS);
    ux_momentum_flux<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, uF_x, rho_u, rho, F_x, edgedx, edgedy, celldx, celldy);
    handle_boundary(nx, ny, mesh, uF_x, NO_INVERT, PACK);

    nthreads_per_block = ceil((nx+1)*ny/(double)NBLOCKS);
    advect_rho_u_and_u_in_x<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, tt, mesh, dt_h, dt, u, v, uF_x, uF_y, 
        vF_x, vF_y, rho_u, rho_v, rho, F_x, F_y, edgedx, edgedy, celldx, celldy);
    handle_boundary(nx+1, ny, mesh, u, INVERT_X, PACK);

    nthreads_per_block = ceil((nx+1)*(ny+1)/(double)NBLOCKS);
    uy_momentum_flux<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, uF_y, rho_u, rho, F_y, edgedx, edgedy, celldx, celldy);
    handle_boundary(nx+1, ny+1, mesh, uF_y, NO_INVERT, PACK);

    nthreads_per_block = ceil((nx+1)*ny/(double)NBLOCKS);
    advect_rho_u_in_y<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, tt, mesh, dt_h, dt, u, v, uF_x, uF_y, vF_x, vF_y, rho_u, rho_v, 
        rho, F_x, F_y, edgedx, edgedy, celldx, celldy);

    nthreads_per_block = ceil((nx+1)*(ny+1)/(double)NBLOCKS);
    vx_momentum_flux<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, vF_x, rho_v, rho, F_x, edgedx, edgedy, celldx, celldy);
    handle_boundary(nx+1, ny+1, mesh, vF_x, NO_INVERT, PACK);

    nthreads_per_block = ceil(nx*(ny+1)/(double)NBLOCKS);
    advect_rho_v_and_v_in_x<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, vF_x, rho_v, rho, F_x, 
        edgedx, edgedy, celldx, celldy);
    handle_boundary(nx, ny+1, mesh, v, INVERT_Y, PACK);

    nthreads_per_block = ceil(nx*ny/(double)NBLOCKS);
    vy_momentum_flux<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, vF_y, rho_v, rho, F_y, edgedx, edgedy, celldx, celldy);
    handle_boundary(nx, ny, mesh, vF_y, NO_INVERT, PACK);

    nthreads_per_block = ceil(nx*(ny+1)/(double)NBLOCKS);
    advect_rho_v_in_y<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, vF_y, rho_v, rho, F_y, 
        edgedx, edgedy, celldx, celldy);
  }
  else {
    nthreads_per_block = ceil((nx+1)*(ny+1)/(double)NBLOCKS);
    uy_momentum_flux<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, uF_y, rho_u, rho, F_y, edgedx, edgedy, celldx, celldy);
    handle_boundary(nx+1, ny+1, mesh, uF_y, NO_INVERT, PACK);

    nthreads_per_block = ceil((nx+1)*ny/(double)NBLOCKS);
    advect_rho_u_and_u_in_y<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, tt, mesh, dt_h, dt, u, v, uF_x, uF_y, vF_x, vF_y, rho_u, 
        rho_v, rho, F_x, F_y, edgedx, edgedy, celldx, celldy);
    handle_boundary(nx+1, ny, mesh, u, INVERT_X, PACK);

    nthreads_per_block = ceil(nx*ny/(double)NBLOCKS);
    ux_momentum_flux<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, uF_x, rho_u, rho, 
        F_x, edgedx, edgedy, celldx, celldy);
    handle_boundary(nx, ny, mesh, uF_x, NO_INVERT, PACK);

    nthreads_per_block = ceil((nx+1)*ny/(double)NBLOCKS);
    advect_rho_u_in_y<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, tt, mesh, dt_h, dt, u, v, uF_x, uF_y, vF_x, vF_y, rho_u, rho_v, 
        rho, F_x, F_y, edgedx, edgedy, celldx, celldy);

    nthreads_per_block = ceil(nx*ny/(double)NBLOCKS);
    vy_momentum_flux<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, vF_y, rho_v, rho, F_y, edgedx, edgedy, celldx, celldy);
    handle_boundary(nx, ny, mesh, vF_y, NO_INVERT, PACK);

    nthreads_per_block = ceil(nx*(ny+1)/(double)NBLOCKS);
    advect_rho_v_and_v_in_y<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, vF_y, rho_v, rho, F_y, 
        edgedx, edgedy, celldx, celldy);
    handle_boundary(nx, ny+1, mesh, v, INVERT_Y, PACK);

    nthreads_per_block = ceil((nx+1)*(ny+1)/(double)NBLOCKS);
    vx_momentum_flux<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, vF_x, rho_v, rho, F_x, edgedx, edgedy, celldx, celldy);

    nthreads_per_block = ceil(nx*(ny+1)/(double)NBLOCKS);
    advect_rho_v_in_x<<<nthreads_per_block, NBLOCKS>>>(
        nx, ny, mesh, dt_h, dt, u, v, vF_x, rho_v, rho, 
        F_x, edgedx, edgedy, celldx, celldy)
  }
}


// Prints some conservation values
void print_conservation(
    const int nx, const int ny, double* rho, double* e, Mesh* mesh) 
{
  double mass_tot = 0.0;
  double energy_tot = 0.0;
#pragma omp parallel for reduction(+:mass_tot, energy_tot)
  for(int ii = PAD; ii < ny-PAD; ++ii) {
    for(int jj = PAD; jj < nx-PAD; ++jj) {
      mass_tot += rho[ind0];
      energy_tot += rho[ind0]*e[ind0];
    }
  }

  double global_mass_tot = mass_tot;
  double global_energy_tot = energy_tot;

#ifdef MPI
  MPI_Reduce(&mass_tot, &global_mass_tot, 1, MPI_DOUBLE, MPI_SUM, MASTER, MPI_COMM_WORLD);
  MPI_Reduce(&energy_tot, &global_energy_tot, 1, MPI_DOUBLE, MPI_SUM, MASTER, MPI_COMM_WORLD);
#endif

  if(mesh->rank == MASTER) {
    printf("total mass: %.12e\n", global_mass_tot);
    printf("total energy: %.12e\n", global_energy_tot);
  }
}

// http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
  template <unsigned int block_size>
__device__ void min_reduce(
    double* data, double* result)
{
  __shared__ sdata[block_size];
  const int ind = blockIdx.x*block_size + threadIdx.x;
  sdata[threadIdx.x] = data[ind];
  __syncthreads();

  min_reduce_in_shared(threadIdx.x, sdata);
  if (threadIdx.x == 0) result[blockIdx.x] = sdata[0];
}

  template <unsigned int block_size>
__device__ void min_reduce_in_shared(
    const int tid, __shared__ double sdata)
{
  if (block_size >= 512) { 
    if (tid < 256) { sdata[tid] = min(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
  if (block_size >= 256) { 
    if (tid < 128) { sdata[tid] = min(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
  if (block_size >= 128) { 
    if (tid < 64) { sdata[tid] = min(sdata[tid], sdata[tid + 64]); } __syncthreads(); }
  if (tid < 32) {
    if (block_size >= 64) sdata[tid] = min(sdata[tid], sdata[tid + 32]);
    if (block_size >= 32) sdata[tid] = min(sdata[tid], sdata[tid + 16]);
    if (block_size >= 16) sdata[tid] = min(sdata[tid], sdata[tid + 8]);
    if (block_size >= 8) sdata[tid] = min(sdata[tid], sdata[tid + 4]);
    if (block_size >= 4) sdata[tid] = min(sdata[tid], sdata[tid + 2]);
    if (block_size >= 2) sdata[tid] = min(sdata[tid], sdata[tid + 1]);
  }
}

