#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "flow.h"
#include "flow.k"
#include "../../comms.h"

// Solve a single timestep on the given mesh
void solve_hydro_2d(
    Mesh* mesh, int tt, double* P, double* rho, double* rho_old, 
    double* e, double* u, double* v, double* rho_u, double* rho_v, 
    double* Qxx, double* Qyy, double* F_x, double* F_y, double* uF_x, 
    double* uF_y, double* vF_x, double* vF_y, double* reduce_array)
{
  if(mesh->rank == MASTER) {
    printf("Timestep:        %.12e\n", mesh->dt, mesh->dt_h);
  }

  int nblocks = ceil(mesh->local_nx*mesh->local_ny/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  equation_of_state<<<nblocks, NTHREADS>>>(
      mesh->local_nx, mesh->local_ny, P, rho, e);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "equation_of_state");

  nblocks = ceil((mesh->local_nx+1)*(mesh->local_ny+1)/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  pressure_acceleration<<<nblocks, NTHREADS>>>(
      mesh->local_nx, mesh->local_ny, mesh->pad, mesh->dt, rho_u, rho_v, 
      u, v, P, rho, mesh->edgedx, mesh->edgedy, 
      mesh->celldx, mesh->celldy);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "pressure_acceleration");

  handle_boundary_2d(mesh->local_nx+1, mesh->local_ny, mesh, u, INVERT_X, PACK);
  handle_boundary_2d(mesh->local_nx, mesh->local_ny+1, mesh, v, INVERT_Y, PACK);

  artificial_viscosity(
      mesh->local_nx, mesh->local_ny, mesh, mesh->dt, Qxx, Qyy, 
      u, v, rho_u, rho_v, rho, 
      mesh->edgedx, mesh->edgedy, mesh->celldx, mesh->celldy);
  gpu_check(hipDeviceSynchronize());

  nblocks = ceil(mesh->local_nx*mesh->local_ny/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  shock_heating_and_work<<<nblocks, NTHREADS>>>(
      mesh->local_nx, mesh->local_ny, mesh->pad, mesh->dt_h, e, P, u, 
      v, rho, Qxx, Qyy, mesh->celldx, mesh->celldy);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "shock_heating_and_work");

  handle_boundary_2d(mesh->local_nx, mesh->local_ny, mesh, e, NO_INVERT, PACK);

  set_timestep(
      mesh->local_nx, mesh->local_ny, Qxx, Qyy, rho, 
      e, mesh, reduce_array, tt == 0, mesh->celldx, mesh->celldy);

  // Perform advection
  advect_mass_and_energy(
      mesh->local_nx, mesh->local_ny, mesh, tt, mesh->dt, mesh->dt_h, rho, e, rho_old, F_x, F_y, 
      uF_x, uF_y, u, v, mesh->edgedx, mesh->edgedy, mesh->celldx, mesh->celldy);
  gpu_check(hipDeviceSynchronize());

  advect_momentum(
      mesh->local_nx, mesh->local_ny, tt, mesh, mesh->dt_h, mesh->dt, u, v, 
      uF_x, uF_y, vF_x, vF_y, rho_u, rho_v, rho, F_x, F_y, 
      mesh->edgedx, mesh->edgedy, mesh->celldx, mesh->celldy);
  gpu_check(hipDeviceSynchronize());
}

// Calculate the pressure from GAMma law equation of state
void artificial_viscosity(
    const int nx, const int ny, Mesh* mesh, const double dt, double* Qxx, 
    double* Qyy, double* u, double* v, double* rho_u, double* rho_v, const double* rho, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  int nblocks = ceil(nx*ny/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  calc_viscous_stresses<<<nblocks, NTHREADS>>>(
      nx, ny, mesh->pad, dt, Qxx, Qyy, u, v, rho_u, rho_v, rho, 
      edgedx, edgedy, celldx, celldy);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "artificial_viscosity");

  handle_boundary_2d(nx, ny, mesh, Qxx, NO_INVERT, PACK);
  handle_boundary_2d(nx, ny, mesh, Qyy, NO_INVERT, PACK);

  nblocks = ceil((nx+1)*(ny+1)/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  viscous_acceleration<<<nblocks, NTHREADS>>>(
      nx, ny, mesh->pad, dt, Qxx, Qyy, u, v, rho_u, rho_v, rho, 
      edgedx, edgedy, celldx, celldy);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "artificial_viscosity");

  handle_boundary_2d(nx+1, ny, mesh, u, INVERT_X, PACK);
  handle_boundary_2d(nx, ny+1, mesh, v, INVERT_Y, PACK);
}

// Calculates the timestep from the current state
void set_timestep(
    const int nx, const int ny, double* Qxx, double* Qyy, 
    const double* rho, const double* e, Mesh* mesh, double* reduce_array, 
    const int first_step, const double* celldx, const double* celldy)
{
  int nblocks = ceil((nx+1)*(ny+1)/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  calc_min_timestep<<<nblocks, NTHREADS>>>(
      nx, ny, mesh->pad, mesh->max_dt, Qxx, Qyy, rho, e, reduce_array, 
      first_step, celldx, celldy);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "calc_min_timestep");

  START_PROFILING(&comms_profile);
  double local_min_dt;
  finish_min_reduce(nblocks, reduce_array, &local_min_dt);

  // Ensure that the timestep does not jump too far from one step to the next
  double global_min_dt = reduce_all_min(local_min_dt);
  const double final_min_dt = min(global_min_dt, C_M*mesh->dt_h);
  mesh->dt = 0.5*(C_T*final_min_dt + mesh->dt_h);
  mesh->dt_h = (first_step) ? mesh->dt : C_T*final_min_dt;
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&comms_profile, "finish_min_reduce");
}

// Perform advection with monotonicity improvement
void advect_mass_and_energy(
    const int nx, const int ny, Mesh* mesh, const int tt, const double dt,
    const double dt_h, double* rho, double* e, double* rho_old, double* F_x, double* F_y, 
    double* eF_x, double* eF_y, const double* u, const double* v, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  int nblocks = ceil(nx*ny/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  store_old_rho<<<nblocks, NTHREADS>>>(nx, ny, mesh->pad, rho, rho_old);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "store_old_rho");

  if(tt % 2 == 0) {
    mass_and_energy_x_advection(
        nx, ny, 1, mesh, dt, dt_h, rho, rho_old, e, u, F_x, eF_x, 
        celldx, edgedx, celldy, edgedy);
    mass_and_energy_y_advection(
        nx, ny, 0, mesh, dt, dt_h, rho, rho_old, e, v, F_y, eF_y, 
        celldx, edgedx, celldy, edgedy);
  }
  else {
    mass_and_energy_y_advection(
        nx, ny, 1, mesh, dt, dt_h, rho, rho_old, e, v, F_y, eF_y, 
        celldx, edgedx, celldy, edgedy);
    mass_and_energy_x_advection(
        nx, ny, 0, mesh, dt, dt_h, rho, rho_old, e, u, F_x, eF_x, 
        celldx, edgedx, celldy, edgedy);
  }
}

// Advect energy and mass in the x direction
void mass_and_energy_x_advection(
    const int nx, const int ny, const int first, Mesh* mesh, const double dt, 
    const double dt_h, double* rho, double* rho_old, double* e, const double* u, 
    double* F_x, double* eF_x, const double* celldx, const double* edgedx, 
    const double* celldy, const double* edgedy)
{
  int nblocks = ceil((nx+1)*ny/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  calc_x_mass_and_energy_flux<<<nblocks, NTHREADS>>>(
      nx, ny, first, mesh->pad, dt, dt_h, rho, rho_old, e, u, 
      F_x, eF_x, celldx, edgedx, celldy, edgedy);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "advect_mass_and_energy");

  handle_boundary_2d(nx+1, ny, mesh, F_x, INVERT_X, PACK);

  nblocks = ceil(nx*ny/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  advect_mass_and_energy_in_x<<<nblocks, NTHREADS>>>(
      nx, ny, first, mesh->pad, dt, dt_h, rho, rho_old, e, u, 
      F_x, eF_x, celldx, edgedx, celldy, edgedy);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "advect_mass_and_energy");

  handle_boundary_2d(nx, ny, mesh, rho, NO_INVERT, PACK);
  handle_boundary_2d(nx, ny, mesh, e, NO_INVERT, PACK);
}

// Advect energy and mass in the y direction
void mass_and_energy_y_advection(
    const int nx, const int ny, const int first, Mesh* mesh, const double dt,
    const double dt_h, double* rho, double* rho_old, double* e, const double* v, 
    double* F_y, double* eF_y, const double* celldx, const double* edgedx, 
    const double* celldy, const double* edgedy)
{
  int nblocks = ceil(nx*(ny+1)/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  calc_y_mass_and_energy_flux<<<nblocks, NTHREADS>>>(
      nx, ny, first, mesh->pad, dt, dt_h, rho, rho_old, e, v, 
      F_y, eF_y, celldx, edgedx, celldy, edgedy);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "advect_mass_and_energy");

  handle_boundary_2d(nx, ny+1, mesh, F_y, INVERT_Y, PACK);

  nblocks = ceil(nx*ny/(double)NTHREADS);
  START_PROFILING(&compute_profile);
  advect_mass_and_energy_in_y<<<nblocks, NTHREADS>>>(
      nx, ny, first, mesh->pad, dt, dt_h, rho, rho_old, e, v, 
      F_y, eF_y, celldx, edgedx, celldy, edgedy);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, "advect_mass_and_energy");

  handle_boundary_2d(nx, ny, mesh, rho, NO_INVERT, PACK);
  handle_boundary_2d(nx, ny, mesh, e, NO_INVERT, PACK);
}

// Advect momentum according to the velocity
void advect_momentum(
    const int nx, const int ny, const int tt, Mesh* mesh, const double dt_h, 
    const double dt, double* u, double* v, double* uF_x, double* uF_y, 
    double* vF_x, double* vF_y, double* rho_u, double* rho_v, 
    const double* rho, const double* F_x, const double* F_y, 
    const double* edgedx, const double* edgedy, const double* celldx, const double* celldy)
{
  int nblocks = 0;
  if(tt % 2) {
    nblocks = ceil(nx*ny/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    ux_momentum_flux<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, uF_x, rho_u, rho, F_x, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx, ny, mesh, uF_x, NO_INVERT, PACK);

    nblocks = ceil((nx+1)*ny/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    advect_rho_u_and_u_in_x<<<nblocks, NTHREADS>>>(
        nx, ny, tt, mesh->pad, dt_h, dt, u, v, uF_x, uF_y, 
        vF_x, vF_y, rho_u, rho_v, rho, F_x, F_y, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx+1, ny, mesh, u, INVERT_X, PACK);

    nblocks = ceil((nx+1)*(ny+1)/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    uy_momentum_flux<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, uF_y, rho_u, rho, F_y, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx+1, ny+1, mesh, uF_y, NO_INVERT, PACK);

    nblocks = ceil((nx+1)*ny/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    advect_rho_u_in_y<<<nblocks, NTHREADS>>>(
        nx, ny, tt, mesh->pad, dt_h, dt, u, v, uF_x, uF_y, vF_x, vF_y, rho_u, rho_v, 
        rho, F_x, F_y, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    nblocks = ceil((nx+1)*(ny+1)/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    vx_momentum_flux<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, vF_x, rho_v, rho, F_x, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx+1, ny+1, mesh, vF_x, NO_INVERT, PACK);

    nblocks = ceil(nx*(ny+1)/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    advect_rho_v_and_v_in_x<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, vF_x, rho_v, rho, F_x, 
        edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx, ny+1, mesh, v, INVERT_Y, PACK);

    nblocks = ceil(nx*ny/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    vy_momentum_flux<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, vF_y, rho_v, rho, F_y, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx, ny, mesh, vF_y, NO_INVERT, PACK);

    nblocks = ceil(nx*(ny+1)/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    advect_rho_v_in_y<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, vF_y, rho_v, rho, F_y, 
        edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");
  }
  else {
    nblocks = ceil((nx+1)*(ny+1)/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    uy_momentum_flux<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, uF_y, rho_u, rho, F_y, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx+1, ny+1, mesh, uF_y, NO_INVERT, PACK);

    nblocks = ceil((nx+1)*ny/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    advect_rho_u_and_u_in_y<<<nblocks, NTHREADS>>>(
        nx, ny, tt, mesh->pad, dt_h, dt, u, v, uF_x, uF_y, vF_x, vF_y, rho_u, 
        rho_v, rho, F_x, F_y, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx+1, ny, mesh, u, INVERT_X, PACK);

    nblocks = ceil(nx*ny/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    ux_momentum_flux<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, uF_x, rho_u, rho, 
        F_x, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx, ny, mesh, uF_x, NO_INVERT, PACK);

    nblocks = ceil((nx+1)*ny/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    advect_rho_u_in_x<<<nblocks, NTHREADS>>>(
        nx, ny, tt, mesh->pad, dt_h, dt, u, v, uF_x, uF_y, vF_x, vF_y, rho_u, rho_v, 
        rho, F_x, F_y, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    nblocks = ceil(nx*ny/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    vy_momentum_flux<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, vF_y, rho_v, rho, F_y, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx, ny, mesh, vF_y, NO_INVERT, PACK);

    nblocks = ceil(nx*(ny+1)/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    advect_rho_v_and_v_in_y<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, vF_y, rho_v, rho, F_y, 
        edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx, ny+1, mesh, v, INVERT_Y, PACK);

    nblocks = ceil((nx+1)*(ny+1)/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    vx_momentum_flux<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, vF_x, rho_v, rho, F_x, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");

    handle_boundary_2d(nx+1, ny+1, mesh, vF_x, NO_INVERT, PACK);

    nblocks = ceil(nx*(ny+1)/(double)NTHREADS);
    START_PROFILING(&compute_profile);
    advect_rho_v_in_x<<<nblocks, NTHREADS>>>(
        nx, ny, mesh->pad, dt_h, dt, u, v, vF_x, rho_v, rho, 
        F_x, edgedx, edgedy, celldx, celldy);
    gpu_check(hipDeviceSynchronize());
    STOP_PROFILING(&compute_profile, "advect momentum");
  }
}

// Prints some conservation values
void print_conservation(
    const int nx, const int ny, double* rho, double* e, double* reduce_array, Mesh* mesh) 
{
  START_PROFILING(&compute_profile);
  int nblocks = ceil(nx*ny/(double)NTHREADS);
  calc_mass_sum<<<nblocks, NTHREADS>>>( 
      nx, ny, mesh->pad, rho, reduce_array);
  gpu_check(hipDeviceSynchronize());
  STOP_PROFILING(&compute_profile, __func__);

  START_PROFILING(&comms_profile);
  double local_mass_tot = 0.0;
  finish_sum_reduce(nblocks, reduce_array, &local_mass_tot);

  double global_mass_tot = reduce_to_master(local_mass_tot);

  if(mesh->rank == MASTER) {
    printf("Total mass:    %.12e\n", global_mass_tot);
  }
  STOP_PROFILING(&comms_profile, "finish_sum_reduce");
}

